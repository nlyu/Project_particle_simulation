#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

int bins_per_side;
int n_bins;

extern double size;

struct bin_t {
    int particles[32];
    int n_particles;
};

__host__ int bin_of_particle(particle_t &particle) {
    double sidelength = size / bins_per_side;
    int b_row = (int)(particle.x / sidelength);
    int b_col = (int)(particle.y / sidelength);
    return b_row + b_col * bins_per_side;
}

__device__ int bin_of_particle_gpu(particle_t &particle, double d_size, int d_bins_per_side) {
    double sidelength = d_size / d_bins_per_side;
    int b_row = (int)(particle.x / sidelength);
    int b_col = (int)(particle.y / sidelength);
    return b_row + b_col * d_bins_per_side;
}

__host__ void init_bins(int n, particle_t *particles,
                  bin_t *d_bins) {
    // Create bins on host
    bin_t *bins = new bin_t[n_bins];
    for (int b = 0; b < n_bins; b++) {
        bins[b].n_particles = 0;
    }
    // Assign each particle to a bin
    for (int k = 0; k < n; k++) {
        int b_idx = bin_of_particle(particles[k]);
        bins[b_idx].particles[bins[b_idx].n_particles++] = k;
    }
    // Copy host bins to device
    hipMemcpy(d_bins, bins, n_bins * sizeof(bin_t), hipMemcpyHostToDevice);

    delete[] bins;
}

__host__ void init_bins_id(int n, particle_t *particles,
                  int * d_bins_id) {
    // Create bins on host
    int * bins_id = new int[n];
    for (int p = 0; p < n; p++) {
        bins_id[p] = bin_of_particle(particles[p]);
    }

    // Copy host bins to device
    hipMemcpy(d_bins_id, bins_id, n * sizeof(int), hipMemcpyHostToDevice);

    delete[] bins_id;
}

//
//  benchmarking program
//
__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if( r2 > cutoff * cutoff )
        return;
    //r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
    double r = sqrt( r2 );

    //
    //  very simple short-range repulsive force
    //
    double coef = ( 1 - cutoff / r ) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t *particles,
                                   bin_t *d_bins,
                                   int *d_bins_id,
                                   int d_n_bins, int d_bins_per_side) {
    // Get thread (bin) ID
    int b1 = threadIdx.x + blockIdx.x * blockDim.x;
    if (b1 >= d_n_bins) return;

    int b1_row = b1 % d_bins_per_side;
    int b1_col = b1 / d_bins_per_side;

    for (int p1 = 0; p1 < d_bins[b1].n_particles; p1++) {
        particles[d_bins[b1].particles[p1]].ax = particles[d_bins[b1].particles[p1]].ay = 0;
    }

    for (int b2_row = max(0, b1_row - 1);
         b2_row <= min(d_bins_per_side - 1, b1_row + 1);
         b2_row++) {
        for (int b2_col = max(0, b1_col - 1);
             b2_col <= min(d_bins_per_side - 1, b1_col + 1);
             b2_col++) {
            int b2 = b2_row + b2_col * d_bins_per_side;
            for (int p1 = 0; p1 < d_bins[b1].n_particles; p1++) {
                for (int p2 = 0; p2 < d_bins[b2].n_particles; p2++) {
                    apply_force_gpu(particles[d_bins[b1].particles[p1]],
                                    particles[d_bins[b2].particles[p2]]);
                }
            }
        }
    }

    // Clear staying and leaving from previous iteration in preparation for move_gpu_step1
    // _bins[b1].n_staying = d_bins[b1].n_leaving = 0;
}

__device__ void move_particle_gpu(particle_t &p, double d_size) {
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p.vx += p.ax * dt;
    p.vy += p.ay * dt;
    p.x  += p.vx * dt;
    p.y  += p.vy * dt;

    //
    //  bounce from walls
    //
    while( p.x < 0 || p.x > d_size )
    {
        p.x  = p.x < 0 ? -(p.x) : 2*d_size-p.x;
        p.vx = -(p.vx);
    }
    while( p.y < 0 || p.y > d_size )
    {
        p.y  = p.y < 0 ? -(p.y) : 2*d_size-p.y;
        p.vy = -(p.vy);
    }
}

__global__ void move_gpu_my1 (particle_t *particles,
                                bin_t *d_bins,
                                double d_size,
                                int *d_bins_id, int d_bins_per_side, int d_n_bins) {
    // Get thread (bin) ID
    int b = threadIdx.x + blockIdx.x * blockDim.x;
    if (b >= d_n_bins) return;

    // Move this bin's particles to either leaving or staying
    for (int p1 = 0, p_id = 0; p1 < d_bins[b].n_particles; p1++) {
        p_id = d_bins[b].particles[p1];
        particle_t &p = particles[p_id];
        move_particle_gpu(p, d_size);
        int new_b_idx = bin_of_particle_gpu(p, d_size, d_bins_per_side);
        if (new_b_idx != b) {
            d_bins_id[p_id] = new_b_idx;
        }
    }
}

__global__ void binning (particle_t *particles,
                         bin_t *d_bins, int * d_bins_id, int d_n){
    int b = threadIdx.x + blockIdx.x * blockDim.x;

    d_bins[b].n_particles = 0;
    for (int p = 0; p < d_n; p++) {
        if(d_bins_id[p] == b){
            d_bins[b].particles[d_bins[b].n_particles++] = p;
        }
    }
}

//
// __global__ void move_gpu_step1 (particle_t *particles,
//                                 bin_t *d_bins,
//                                 double d_size, int d_bins_per_side, int d_n_bins) {
//     // Get thread (bin) ID
//     int b = threadIdx.x + blockIdx.x * blockDim.x;
//     if (b >= d_n_bins) return;
//
//     // Move this bin's particles to either leaving or staying
//     for (int p1 = 0; p1 < d_bins[b].n_particles; p1++) {
//         particle_t &p = particles[d_bins[b].particles[p1]];
//         move_particle_gpu(p, d_size);
//         int new_b_idx = bin_of_particle_gpu(p, d_size, d_bins_per_side);
//         if (new_b_idx != b) {
//             d_bins[b].leaving[d_bins[b].n_leaving++] = d_bins[b].particles[p1];
//         } else {
//             d_bins[b].staying[d_bins[b].n_staying++] = d_bins[b].particles[p1];
//         }
//     }
//     assert(d_bins[b].n_leaving < 32);
//     assert(d_bins[b].n_staying < 32);
// }
//
// __global__ void move_gpu_step2 (particle_t *particles,
//                                 bin_t *d_bins,
//                                 double d_size, int d_bins_per_side, int d_n_bins) {
//     // Get thread (bin) ID
//     int b = threadIdx.x + blockIdx.x * blockDim.x;
//     if (b >= d_n_bins) return;
//
//     // Consolidate staying and particles from neighbor bins' leaving
//     // lists. Assumes particles don't go so fast that they jump over bins.
//     for (int p1 = 0; p1 < d_bins[b].n_staying; p1++) {
//         d_bins[b].particles[p1] = d_bins[b].staying[p1];
//     }
//     d_bins[b].n_particles = d_bins[b].n_staying;
//
//     int b1_row = b % d_bins_per_side;
//     int b1_col = b / d_bins_per_side;
//     for (int b2_row = max(0, b1_row - 1);
//          b2_row <= min(d_bins_per_side - 1, b1_row + 1);
//          b2_row++) {
//         for (int b2_col = max(0, b1_col - 1);
//              b2_col <= min(d_bins_per_side - 1, b1_col + 1);
//              b2_col++) {
//             int b2 = b2_row + b2_col * d_bins_per_side;
//             for (int p2 = 0; p2 < d_bins[b2].n_leaving; p2++) {
//                 particle_t &p = particles[d_bins[b2].leaving[p2]];
//                 int new_b_idx = bin_of_particle_gpu(p, d_size, d_bins_per_side);
//                 if (new_b_idx == b) {
//                     d_bins[b].particles[d_bins[b].n_particles++] = d_bins[b2].leaving[p2];
//                 }
//             }
//         }
//     }
//     assert(d_bins[b].n_particles < 32);
// }



int main( int argc, char **argv )
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }

    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    // Initialize bins
    bins_per_side = read_int(argc, argv, "-b", size / (0.01*3));
    n_bins = bins_per_side * bins_per_side;
    bin_t *d_bins;
    hipMalloc((void **) &d_bins, n_bins * sizeof(bin_t));
    init_bins(n, particles, d_bins);

    int * d_bins_id;
    hipMalloc((void **) &d_bins_id, n * sizeof(int));
    init_bins_id(n, particles, d_bins_id);


    hipDeviceSynchronize();   // Block until all preceeding tasks on all threads are done
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

        int blks = (n_bins + NUM_THREADS - 1) / NUM_THREADS;
        compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, d_bins, d_bins_id,
                                                      n_bins, bins_per_side);

        //
        //  move particles
        //

        //move_gpu_step1 <<< blks, NUM_THREADS >>> (d_particles, d_bins, size, bins_per_side, n_bins);
        //move_gpu_step2 <<< blks, NUM_THREADS >>> (d_particles, d_bins, size, bins_per_side, n_bins);
        move_gpu_my1 <<< blks, NUM_THREADS >>> (d_particles, d_bins, size, d_bins_id, bins_per_side, n_bins, n);
        binning <<< blks, NUM_THREADS >>> (d_particles, d_bins, d_bins_id, n);
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
            // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
        }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;

    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

    free( particles );
    hipFree(d_particles);
    hipFree(d_bins);
    if( fsave )
        fclose( fsave );

    return 0;
}
